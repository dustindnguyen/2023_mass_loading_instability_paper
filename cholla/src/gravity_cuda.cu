
#include <hip/hip_runtime.h>
/*! \file gravity_cuda.cu
 *  \brief Definitions of functions to calculate gravitational
           acceleration in 1, 2, and 3D. Called in Update_Conserved_Variables
           functions in hydro_cuda.cu. */
#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"gravity_cuda.h"

__device__ void calc_g_1D(int xid, int x_off, int n_ghost, Real dx, Real xbound, Real *gx)
{
  Real x_pos, r_disk, r_halo;
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;

  // for disk components, calculate polar r
  //r_disk = 0.220970869121;
  //r_disk = 6.85009694274;
  r_disk = 13.9211647546;
  //r_disk = 20.9922325665;
  // for halo, calculate spherical r
  r_halo = sqrt(x_pos*x_pos + r_disk*r_disk);

  // set properties of halo and disk (these must match initial conditions)
  Real a_disk_z, a_halo, M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20.0; // halo concentration
  R_h = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  z_d = 3.5/5.0; // disk scale height in kpc
  phi_0_h = GN * M_h / (log(1.0+c_vir) - c_vir / (1.0+c_vir));
  x = r_halo / R_h;
  
  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo = - phi_0_h * (log(1+x) - x/(1+x)) / (r_halo*r_halo);
  a_disk_z = - GN * M_d * x_pos * (R_d + sqrt(x_pos*x_pos + z_d*z_d)) / ( pow(r_disk*r_disk + pow(R_d + sqrt(x_pos*x_pos + z_d*z_d), 2), 1.5) * sqrt(x_pos*x_pos + z_d*z_d) );

  // total acceleration is the sum of the halo + disk components
  *gx = (x_pos/r_halo)*a_halo + a_disk_z;

  return;

}


__device__ void calc_g_2D(int xid, int yid, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real *gx, Real *gy)
{
  Real x_pos, y_pos, r, phi;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;

  // for Gresho, also need r & phi
  r = sqrt(x_pos*x_pos + y_pos*y_pos);
  phi = atan2(y_pos, x_pos);

/*
  // set acceleration to balance v_phi in Gresho problem
  if (r < 0.2) {
    *gx = -cos(phi)*25.0*r;
    *gy = -sin(phi)*25.0*r;
  }
  else if (r >= 0.2 && r < 0.4) {
    *gx = -cos(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
    *gy = -sin(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
  }
  else {
    *gx = 0.0;
    *gy = 0.0;
  }
*/
/*
  // set gravitational acceleration for Keplarian potential
  Real M;
  M = 1*Msun;
  *gx = -cos(phi)*GN*M/(r*r);
  *gy = -sin(phi)*GN*M/(r*r);
*/
  // set gravitational acceleration for Kuzmin disk + NFW halo
  Real a_d, a_h, a, M_vir, M_d, R_vir, R_d, R_s, M_h, c_vir, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun (assume all gas)
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20; // halo concentration
  R_s = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  
  // calculate acceleration
  x = r / R_s;
  a_d = GN * M_d * r * pow(r*r + R_d*R_d, -1.5);
  a_h = GN * M_h * (log(1+x)- x / (1+x)) / ((log(1+c_vir) - c_vir / (1+c_vir)) * r*r);
  a = a_d + a_h;

  *gx = -cos(phi)*a;
  *gy = -sin(phi)*a;

  return;
}


__device__ void calc_g_3D(int xid, int yid, int zid, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real *gx, Real *gy, Real *gz)
{
  Real x_pos, y_pos, z_pos;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;
  z_pos = (z_off + zid - n_ghost + 0.5)*dz + zbound;
  
  // Calculate the centred positions, index C refers to center shifted positions 
  Real center_x, center_y, center_z;  // centres defined in code units [kpc]
  Real x_pos_C, y_pos_C, z_pos_C;
  center_x = 3.0;
  center_y = 3.0;
  center_z = 3.0; 
  x_pos_C  = x_pos - center_x;
  y_pos_C  = y_pos - center_y; 
  z_pos_C  = z_pos - center_z; 

  Real r_sis, theta_sis, phi_sis;
  Real sigma_sis, a_sis, R_sis_core;
  // for sis (singular isothermal sphere), calculate spherical coordinates in code units  
  r_sis     = sqrt(x_pos_C*x_pos_C + y_pos_C*y_pos_C + z_pos_C*z_pos_C);     
  theta_sis = atan2(sqrt(x_pos_C*x_pos_C+y_pos_C*y_pos_C), z_pos_C);
  phi_sis   = atan2(y_pos_C, x_pos_C);

  // set the properties of singular isothermal sphere 
  // velocity dispersion is defined in cgs units (km/s with 1e5 for cm/s) then converted to code units
  // R_sis_core describes the core of the isothermal sphere, this is to avoid the divergence at r->0 ; this is done in code units
  // for 256^3 , (2kpc)^3, each cell is of order 7.8 pc ; so lets set the core to 50 pc. 
  sigma_sis = 200.0 * 1.0e5  / (VELOCITY_UNIT);  // velocity dispersion 
  R_sis_core = 0.1;
  
  // calculate the acceleration 
  a_sis = - 2.0 * sigma_sis*sigma_sis / (r_sis + R_sis_core) ; 
  
  // total acceleration is the sum of the halo + disk components
  *gx = a_sis * cos(phi_sis) * sin(theta_sis);
  *gy = a_sis * sin(phi_sis) * sin(theta_sis);
  *gz = a_sis * cos(theta_sis);

  return;
}

#endif //CUDA

